#include "hip/hip_runtime.h"
/*
 *  k-step FM-index (benchmarking for CPU and GPU)
 *  Copyright (c) 2011-2017 by Alejandro Chacon  <alejandro.chacond@gmail.com>
 *
 *  This program is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 * PROJECT: k-step FM-index (benchmarking for CPU and GPU)
 * AUTHOR(S): Alejandro Chacon <alejandro.chacond@gmail.com>
 */


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#if defined(K_STEPS) || defined(NUM_BITMAPS) || defined(NUM_COUNTERS)
#else
  #define K_STEPS         2
  #define NUM_CHUNK       64
  #define NUM_COUNTERS    16
#endif

#if defined(CUDA_NUM_THREADS) || defined(MAX_THREADS_PER_SM) || defined(DEVICE)
#else
  #define CUDA_NUM_THREADS      128
  #define MAX_THREADS_PER_SM    128
  #define DEVICE                0
#endif

#define NUM_BITMAPS               (NUM_CHUNK / 32)
#define BITS_PER_BASE             2

#define SIZE_WARP                 32
#define NUM_WARPS_PER_BLOCK       (1 + ((CUDA_NUM_THREADS - 1) / SIZE_WARP))

#define SIZE_VECTOR_TYPE          4
#define TOTAL_NUM_BITMAPS         ( BITS_PER_BASE * NUM_BITMAPS * K_STEPS )

#define NUM_COUNTERS_SLIM         ( NUM_COUNTERS / 2 )
#define SIZE_ENTRY                ( TOTAL_NUM_BITMAPS + NUM_COUNTERS_SLIM )
#define THREADS_PER_REQUEST       ( SIZE_ENTRY / SIZE_VECTOR_TYPE )
#define REQUESTS_PER_WARP         ( SIZE_WARP / THREADS_PER_REQUEST )

#define BYTES_PER_LOAD            16
#define LOADS_PER_WARP            32
#define BYTES_PER_WARP            ( BYTES_PER_LOAD * LOADS_PER_WARP )

// Note: input queries have to be multiple of warp size
#define NUM_LF_MAPPINGS           32
#define NUM_LOADS                 ( NUM_LF_MAPPINGS / REQUESTS_PER_WARP )

typedef struct {
  uint32_t data[SIZE_ENTRY];
} bitcnt_t;

typedef struct {
  uint32_t steps;
  uint32_t bwtsize;
  uint32_t ncounters;
  uint32_t nentries;
  uint32_t chunk;
  uint32_t nbitmaps;
  uint32_t *h_dollarPositionBWT;
  uint32_t *h_dollarBaseBWT;
  uint32_t *h_modposdollarBWT;
  bitcnt_t *h_index;
  uint32_t *d_dollarPositionBWT;
  uint32_t *d_dollarBaseBWT;
  uint32_t *d_modposdollarBWT;
  bitcnt_t *d_index;
} fmi_t;


extern "C"
static void HandleError( hipError_t err, const char *file,  int32_t line ) {
  if (err != hipSuccess) {
    printf( "%s in %s at line %d\n", hipGetErrorString(err),  file, line );
     exit( EXIT_FAILURE );
  }
}

/* static __device__ __inline__ uint4 __ldv(const uint4 *ptr)
{ 
  uint4 ret; 
  asm volatile ("ld.global.cv.v4.u32 {%0, %1, %2, %3}, [%4];" : "=r"(ret.x), "=r"(ret.y), "=r"(ret.z), "=r"(ret.w) : "l"(ptr)); 
  return ret; 
} */

inline __device__ uint32_t countBitmap(uint32_t bitmap, int32_t shift, uint32_t sharedIdxEntry)
{
  uint32_t mask;
  mask = 0xFFFFFFFF << (32 - shift);
  mask = (shift > 32) ? 0xFFFFFFFF : mask;
  mask = (shift > 0) ? mask : 0x0;
  mask = (sharedIdxEntry) ? ~mask : mask;
  return (__popc(bitmap & mask));
}

inline __device__ uint32_t computeBitmaps(uint4 bitmap, uint32_t interval, uint32_t indexBase, uint32_t localRequestThreadIdx, uint32_t sharedIdxEntry)
{
  uint32_t bitmapA;
  uint32_t bit0 = indexBase & 0x01;
  uint32_t bit1 = indexBase & 0x02;
  uint32_t bit2 = indexBase & 0x04;
  uint32_t bit3 = indexBase & 0x08;

  int32_t shift = (interval % NUM_CHUNK) - ((localRequestThreadIdx - 2) * 32);

  bitmap.x = bit0 ? bitmap.x : ~bitmap.x;
  bitmap.y = bit1 ? bitmap.y : ~bitmap.y;

  bitmap.z = bit2 ? bitmap.z : ~bitmap.z;
  bitmap.w = bit3 ? bitmap.w : ~bitmap.w;

  bitmapA = (bitmap.x & bitmap.y) & (bitmap.z & bitmap.w);
  bitmapA = countBitmap(bitmapA, shift, sharedIdxEntry);

  return (bitmapA);
}

inline __device__ uint32_t selectCounter(uint4 counters, uint32_t indexBase)
{
  uint32_t counter;
  if (indexBase == 0) counter = counters.x;
  if (indexBase == 1) counter = counters.y;
  if (indexBase == 2) counter = counters.z;
  if (indexBase == 3) counter = counters.w;
  return(counter);
}

inline __device__ uint32_t reduceEntry(uint32_t resultBitmaps, uint32_t localThreadIdx, uint32_t sharedIdxEntry, uint32_t resultCounters)
{
  uint32_t result;
  for (int32_t i = 1; i < THREADS_PER_REQUEST; i *= 2){
    int32_t n = __shfl_down((int32_t) resultBitmaps, i, 32);
    resultBitmaps += n;
  }
  result = (sharedIdxEntry) ? resultCounters - resultBitmaps : resultCounters + resultBitmaps;
  result = __shfl((int32_t) result, (int32_t) ((localThreadIdx % REQUESTS_PER_WARP) * THREADS_PER_REQUEST));
  return(result);
}

__global__ void searchIndexKernel(uint32_t bwtsize, uint32_t chunk, bitcnt_t *indexFM,
                                  uint32_t *dollarPositionBWT, uint32_t *dollarBaseBWT, uint32_t *modposdollarBWT,
                                  uint32_t numQueries, uint32_t sizeQuery, char *queries, uint32_t *results)
{
  uint32_t *groupQueries = NULL;

  uint32_t indexBase, flg2, flg3;
  uint32_t word, queryWord;
  uint32_t bit0, bit1;
  uint32_t globalThreadIdx, localThreadIdx,
           globalWarpIdx, localRequestThreadIdx, indexData, auxIndexData, result;

  globalThreadIdx = blockIdx.x * MAX_THREADS_PER_SM + threadIdx.x;

  if ((threadIdx.x < MAX_THREADS_PER_SM) && (globalThreadIdx < (numQueries * 2))){
    globalWarpIdx         = globalThreadIdx / SIZE_WARP;
    localThreadIdx        = threadIdx.x % SIZE_WARP;
    localRequestThreadIdx = localThreadIdx % THREADS_PER_REQUEST;
    indexData             = localRequestThreadIdx;

    uint32_t interval = (localThreadIdx % 2) ? bwtsize : 0;
    uint32_t aux_interval, resultBitmaps, idxEntry, resultCounters;

    groupQueries = (uint32_t *) (queries + ((globalWarpIdx / 2) * SIZE_WARP * sizeQuery));
    groupQueries = (globalWarpIdx % 2) ? groupQueries + 16 : groupQueries;

    //#pragma unroll 1
    for(queryWord = 0; queryWord < sizeQuery/4; queryWord++) {
      word = groupQueries[queryWord * SIZE_WARP + (localThreadIdx / 2)] >> 1;
      //#pragma unroll 1
      for(int32_t j = 0; j < 4/K_STEPS; j++) {
        indexBase = 0x0;
        for(int32_t s = 0; s < K_STEPS; s++){
          bit1 = word & 0x02000000;
          flg2 = word & 0x01000000;
          flg3 = flg2 ^ 0x01000000;
          bit0 = bit1 ? flg3 : flg2;
          indexBase |= ((bit1 | bit0) >> (24 - (s * 2)));
          word <<= 8;
        }

        idxEntry = (((((interval / NUM_CHUNK) % 2) != 0) && (indexBase <  NUM_COUNTERS_SLIM)) || 
                    ((((interval / NUM_CHUNK) % 2) == 0) && (indexBase >= NUM_COUNTERS_SLIM))) ? 1 : 0;

        //#pragma unroll 1
        for(int32_t i = 0; i < NUM_LOADS; i++){
          uint32_t  nSharedEntry, sharedInterval, sharedIndexBase, sharedIdxEntry;
          uint4 loadData;

          nSharedEntry = ((REQUESTS_PER_WARP * i) + (localThreadIdx / THREADS_PER_REQUEST));
          sharedInterval  = __shfl((int32_t) interval,  (int32_t) nSharedEntry);
          sharedIndexBase  = __shfl((int32_t) indexBase, (int32_t) nSharedEntry);
          sharedIdxEntry  = __shfl((int32_t) idxEntry,  (int32_t) nSharedEntry);

          auxIndexData = (sharedIdxEntry * THREADS_PER_REQUEST) + ((sharedIndexBase >> 2) & 0x1);
          indexData = (localRequestThreadIdx < 2) ? auxIndexData : indexData;

          #if defined(LDG)
            loadData = __ldg(&(((uint4 *) (indexFM[sharedInterval/NUM_CHUNK].data))[indexData]));
          #else
            loadData = ((uint4 *) (indexFM[sharedInterval/NUM_CHUNK].data))[indexData];
          #endif

          resultBitmaps  = computeBitmaps(loadData, sharedInterval, sharedIndexBase, localRequestThreadIdx, sharedIdxEntry);
          resultCounters = selectCounter(loadData, sharedIndexBase & 0x3);
          resultBitmaps  = (localRequestThreadIdx < 2) ? 0 : resultBitmaps;
          result  = reduceEntry(resultBitmaps, localThreadIdx, sharedIdxEntry, resultCounters);
          aux_interval = (((REQUESTS_PER_WARP * i) <= localThreadIdx) && (localThreadIdx < (REQUESTS_PER_WARP * (i + 1)))) ? result : aux_interval;
        }
        for(int32_t s = 0; s < K_STEPS; s++){
          if(modposdollarBWT[s] == (interval / NUM_CHUNK)){
            aux_interval = ( ~idxEntry && (indexBase == dollarBaseBWT[s]) && (interval > dollarPositionBWT[s])) ? aux_interval - 1 : aux_interval;
            aux_interval = (  idxEntry && (indexBase == dollarBaseBWT[s]) && (interval > dollarPositionBWT[s])) ? aux_interval + 1 : aux_interval;
          }
        }
        interval = aux_interval;
      }
    }
    results[globalThreadIdx] = interval;
  }
}

extern "C"
void searchIndexGPU(void *index, void *dataqueries, void *resIntervals)
{
  fmi_t *fmi = (fmi_t *) index;
  res_t *res=(res_t *) resIntervals;
  qrys_t *qrys=(qrys_t *) dataqueries;
  int32_t blocks=((qrys->num * 2) / MAX_THREADS_PER_SM) + ((qrys->num%MAX_THREADS_PER_SM) ? 1 : 0);
  int32_t threads=CUDA_NUM_THREADS;

  printf("[Num. Entries] real: %u saved: %u, size: %u\n", fmi->bwtsize/fmi->chunk, fmi->nentries, (uint32_t)sizeof(bitcnt_t));
  printf("Blocks: %d - Th_block %d - Th_sm %d\n", blocks, threads, MAX_THREADS_PER_SM);

  searchIndexKernel<<<blocks,threads>>>(fmi->bwtsize, fmi->chunk, fmi->d_index,
                      fmi->d_dollarPositionBWT, fmi->d_dollarBaseBWT,
                      fmi->d_modposdollarBWT, qrys->num, qrys->size,
                      qrys->d_queries, res->d_results);
  hipDeviceSynchronize();
}

extern "C"
int32_t transferCPUtoGPU(void *index, void *dataqueries, void *resIntervals)
{
  fmi_t *fmi = (fmi_t *) index;
  res_t *res = (res_t *) resIntervals;
  qrys_t *qrys = (qrys_t *) dataqueries;

  CUDA_HANDLE_ERROR(hipSetDevice(DEVICE));

  // allocate & transfer FMIndex to GPU
  CUDA_HANDLE_ERROR(hipMalloc((void**)&fmi->d_index, (fmi->nentries)*sizeof(bitcnt_t)));
  CUDA_HANDLE_ERROR(hipMemcpy(fmi->d_index, fmi->h_index, (fmi->nentries)*sizeof(bitcnt_t), hipMemcpyHostToDevice));

  // allocate & transfer dollar positions etc ... IN CONSTANT MEMORY
  CUDA_HANDLE_ERROR(hipMalloc((void**)&fmi->d_dollarPositionBWT, fmi->steps * sizeof(uint32_t)));
  CUDA_HANDLE_ERROR(hipMemcpy(fmi->d_dollarPositionBWT, fmi->h_dollarPositionBWT, fmi->steps * sizeof(uint32_t), hipMemcpyHostToDevice));
  //CUDA_HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_dollarPositionBWT), fmi->h_dollarPositionBWT, fmi->steps * sizeof(uint32_t), 0, hipMemcpyHostToDevice));

  CUDA_HANDLE_ERROR(hipMalloc((void**)&fmi->d_dollarBaseBWT, fmi->steps * sizeof(uint32_t)));
  CUDA_HANDLE_ERROR(hipMemcpy(fmi->d_dollarBaseBWT, fmi->h_dollarBaseBWT, fmi->steps * sizeof(uint32_t), hipMemcpyHostToDevice));
  //CUDA_HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(fmi->d_dollarBaseBWT), fmi->h_dollarBaseBWT, sizeof(uint32_t), 0, hipMemcpyHostToDevice));

  CUDA_HANDLE_ERROR(hipMalloc((void**)&fmi->d_modposdollarBWT, fmi->steps * sizeof(uint32_t)));
  CUDA_HANDLE_ERROR(hipMemcpy(fmi->d_modposdollarBWT, fmi->h_modposdollarBWT, fmi->steps * sizeof(uint32_t), hipMemcpyHostToDevice));
  //CUDA_HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(fmi->d_modposdollarBWT), fmi->h_modposdollarBWT, sizeof(uint32_t), 0, hipMemcpyHostToDevice));

  // allocate & transfer Queries to GPU
  CUDA_HANDLE_ERROR(hipMalloc((void**)&qrys->d_queries, (qrys->num)*(qrys->size)*sizeof(char)));
  CUDA_HANDLE_ERROR(hipMemcpy(qrys->d_queries, qrys->h_queries, (qrys->num)*(qrys->size)*sizeof(char), hipMemcpyHostToDevice));

  // allocate Results
  CUDA_HANDLE_ERROR(hipMalloc((void**)&res->d_results, 2*(res->num)*sizeof(uint32_t)));
  CUDA_HANDLE_ERROR(hipMemset(res->d_results, 0, 2*(res->num)*sizeof(uint32_t)));

  return (SUCCESS);
}

extern "C"
int32_t transferGPUtoCPU(void *resIntervals)
{
  res_t *res = (res_t *) resIntervals;
  CUDA_HANDLE_ERROR(hipMemcpy(res->h_results, res->d_results, 2*(res->num)*sizeof(uint32_t), hipMemcpyDeviceToHost));
  return (SUCCESS);
}

extern "C"
int32_t freeIndexGPU(void **index)
{
  fmi_t *fmi = (fmi_t *) (*index);
  if(fmi->d_dollarPositionBWT != NULL){
    hipFree(fmi->d_dollarPositionBWT);
    fmi->d_dollarPositionBWT=NULL;
  }
  if(fmi->d_dollarBaseBWT != NULL){
    hipFree(fmi->d_dollarBaseBWT);
    fmi->d_dollarBaseBWT=NULL;
  }
  if(fmi->d_modposdollarBWT != NULL){
    hipFree(fmi->d_modposdollarBWT);
    fmi->d_modposdollarBWT=NULL;
  }
  if(fmi->d_index != NULL){
    hipFree(fmi->d_index);
    fmi->d_index=NULL;
  }
  return(SUCCESS);
}

extern "C"
int32_t freeQueriesGPU(void **dataqueries)
{
  qrys_t *qrys = (qrys_t *) (*dataqueries);
  if(qrys->d_queries != NULL){
    hipFree(qrys->d_queries);
    qrys->d_queries=NULL;
  }
  return(SUCCESS);
}

extern "C"
int32_t freeResultsGPU(void **resIntervals)
{
  res_t *res = (res_t *) (*resIntervals);
  if(res->d_results != NULL){
    hipFree(res->d_results);
    res->d_results=NULL;
  }
  return(SUCCESS);
}
